#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<>
#include"sde_builder_cuda.h"
#include<cassert>
#include"random_kernel_initializers.cuh"
#include"mc_types.h"


namespace heston_kernels {


	namespace euler_scheme {

		__global__
			void generatePathsKernelDouble1D(sde_builder_cuda::HestonModel<double> heston, double *d_paths, hiprandState_t* states,
				unsigned int nPaths, unsigned int nSteps, double dt) {
			// Path index
			const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

			if (idx < nPaths) {
				double last = heston.init1();
				double var = heston.init2();
				double last_new{};
				double var_new{};
				double z1{};
				double z2{};
				d_paths[idx] = last;

				unsigned int i = 0;
				for (int k = idx + nPaths; k < nSteps*nPaths; k += nPaths) {
					z1 = hiprand_normal(&states[idx]);
					z2 = hiprand_normal(&states[idx]);
					last_new = last + heston.drift1(i*dt, last, var)*dt +
						heston.diffusion1(i*dt, last, var)*sqrtf(dt)*z1;
					var_new = var + heston.drift2(i*dt, last, var)*dt +
						heston.diffusion2(i*dt, last, var)*sqrtf(dt)*
						(heston.rho() * z1 + sqrt(1.0 - heston.rho()*heston.rho()) * z2);
					d_paths[k] = last_new;
					last = last_new;
					var = var_new;
					i++;
				}
			}
		}

		__global__
			void generatePathsKernelFloat1D(sde_builder_cuda::HestonModel<float> heston, float *d_paths, hiprandState_t* states,
				unsigned int nPaths, unsigned int nSteps, float dt) {
			// Path index
			const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

			if (idx < nPaths) {
				float last = heston.init1();
				float var = heston.init2();
				float last_new{};
				float var_new{};
				float z1{};
				float z2{};
				d_paths[idx] = last;

				unsigned int i = 0;
				for (int k = idx + nPaths; k < nSteps*nPaths; k += nPaths) {
					z1 = hiprand_normal(&states[idx]);
					z2 = hiprand_normal(&states[idx]);
					last_new = last + heston.drift1(i*dt, last, var)*dt +
						heston.diffusion1(i*dt, last, var)*sqrtf(dt)*z1;
					var_new = var + heston.drift2(i*dt, last, var)*dt +
						heston.diffusion2(i*dt, last, var)*sqrtf(dt)*
						(heston.rho() * z1 + sqrt(1.0 - heston.rho()*heston.rho()) * z2);
					d_paths[k] = last_new;
					last = last_new;
					var = var_new;
					i++;
				}
			}
		}

		__global__
			void generatePathsKernelDouble2D(sde_builder_cuda::HestonModel<double> heston, double *d_paths, hiprandState_t* states,
				unsigned int nPathsWidth, unsigned int nPathsHeight, unsigned int nSteps, double dt) {
			// Path index
			const unsigned int c_idx = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int r_idx = blockIdx.y * blockDim.y + threadIdx.y;
			const unsigned int t_idx = c_idx + nPathsWidth * r_idx;
			const unsigned int nPaths = nPathsWidth * nPathsHeight;

			if (t_idx < nPaths) {
				double last = heston.init1();
				double var = heston.init2();
				double last_new{};
				double var_new{};
				double z1{};
				double z2{};
				d_paths[t_idx] = last;

				unsigned int i = 0;
				for (unsigned int k = t_idx + nPaths; k < nSteps*nPaths; k += nPaths) {
					z1 = hiprand_normal(&states[t_idx]);
					z2 = hiprand_normal(&states[t_idx]);
					last_new = last + heston.drift1(i*dt, last, var)*dt +
						heston.diffusion1(i*dt, last, var)*sqrtf(dt)*z1;
					var_new = var + heston.drift2(i*dt, last, var)*dt +
						heston.diffusion2(i*dt, last, var)*sqrtf(dt)*
						(heston.rho() * z1 + sqrt(1.0 - heston.rho()*heston.rho()) * z2);
					d_paths[k] = last_new;
					last = last_new;
					var = var_new;
					i++;
				}
			}
		}

		__global__
			void generatePathsKernelFloat2D(sde_builder_cuda::HestonModel<float> heston, float *d_paths, hiprandState_t* states,
				unsigned int nPathsWidth, unsigned int nPathsHeight, unsigned int nSteps, float dt) {
			// Path index
			const unsigned int c_idx = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int r_idx = blockIdx.y * blockDim.y + threadIdx.y;
			const unsigned int t_idx = c_idx + nPathsWidth * r_idx;
			const unsigned int nPaths = nPathsWidth * nPathsHeight;

			if (t_idx < nPaths) {
				float last = heston.init1();
				float var = heston.init2();
				float last_new{};
				float var_new{};
				float z1{};
				float z2{};
				d_paths[t_idx] = last;

				unsigned int i = 0;
				for (unsigned int k = t_idx + nPaths; k < nSteps*nPaths; k += nPaths) {
					z1 = hiprand_normal(&states[t_idx]);
					z2 = hiprand_normal(&states[t_idx]);
					last_new = last + heston.drift1(i*dt, last, var)*dt +
						heston.diffusion1(i*dt, last, var)*sqrtf(dt)*z1;
					var_new = var + heston.drift2(i*dt, last, var)*dt +
						heston.diffusion2(i*dt, last, var)*sqrtf(dt)*
						(heston.rho() * z1 + sqrt(1.0 - heston.rho()*heston.rho()) * z2);
					d_paths[k] = last_new;
					last = last_new;
					var = var_new;
					i++;
				}
			}
		}


		__global__
			void generatePathsKernelDouble3D(sde_builder_cuda::HestonModel<double> heston, double *d_paths, hiprandState_t* states,
				unsigned int nPathsWidth, unsigned int nPathsHeight, unsigned int nPathsDepth,
				unsigned int nSteps, double dt) {
			// Path index
			const unsigned int c_idx = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int r_idx = blockIdx.y * blockDim.y + threadIdx.y;
			const unsigned int l_idx = blockIdx.z * blockDim.z + threadIdx.z;
			const unsigned int t_idx = c_idx + nPathsWidth * r_idx + nPathsWidth * nPathsHeight*l_idx;
			const unsigned int nPaths = nPathsWidth * nPathsHeight * nPathsDepth;

			if (t_idx < nPaths) {
				double last = heston.init1();
				double var = heston.init2();
				double last_new{};
				double var_new{};
				double z1{};
				double z2{};
				d_paths[t_idx] = last;

				unsigned int i = 0;
				for (unsigned int k = t_idx + nPaths; k < nSteps*nPaths; k += nPaths) {
					z1 = hiprand_normal(&states[t_idx]);
					z2 = hiprand_normal(&states[t_idx]);
					last_new = last + heston.drift1(i*dt, last, var)*dt +
						heston.diffusion1(i*dt, last, var)*sqrtf(dt)*z1;
					var_new = var + heston.drift2(i*dt, last, var)*dt +
						heston.diffusion2(i*dt, last, var)*sqrtf(dt)*
						(heston.rho() * z1 + sqrt(1.0 - heston.rho()*heston.rho()) * z2);
					d_paths[k] = last_new;
					last = last_new;
					var = var_new;
					i++;
				}
			}
		}

		__global__
			void generatePathsKernelFloat3D(sde_builder_cuda::HestonModel<float> heston, float *d_paths, hiprandState_t* states,
				unsigned int nPathsWidth, unsigned int nPathsHeight, unsigned int nPathsDepth,
				unsigned int nSteps, float dt) {
			// Path index
			const unsigned int c_idx = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int r_idx = blockIdx.y * blockDim.y + threadIdx.y;
			const unsigned int l_idx = blockIdx.z * blockDim.z + threadIdx.z;
			const unsigned int t_idx = c_idx + nPathsWidth * r_idx + nPathsWidth * nPathsHeight*l_idx;
			const unsigned int nPaths = nPathsWidth * nPathsHeight * nPathsDepth;

			if (t_idx < nPaths) {
				float last = heston.init1();
				float var = heston.init2();
				float last_new{};
				float var_new{};
				float z1{};
				float z2{};
				d_paths[t_idx] = last;

				unsigned int i = 0;
				for (unsigned int k = t_idx + nPaths; k < nSteps*nPaths; k += nPaths) {
					z1 = hiprand_normal(&states[t_idx]);
					z2 = hiprand_normal(&states[t_idx]);
					last_new = last + heston.drift1(i*dt, last, var)*dt +
						heston.diffusion1(i*dt, last, var)*sqrtf(dt)*z1;
					var_new = var + heston.drift2(i*dt, last, var)*dt +
						heston.diffusion2(i*dt, last, var)*sqrtf(dt)*
						(heston.rho() * z1 + sqrt(1.0 - heston.rho()*heston.rho()) * z2);
					d_paths[k] = last_new;
					last = last_new;
					var = var_new;
					i++;
				}
			}
		}


	};


	namespace milstein_scheme {






	};


};


namespace fdm_engine_cuda {

	using mc_types::FDMScheme;
	using mc_types::GPUConfiguration;
	using mc_types::PathValuesType;

	void fdm_engine_cuda::HestonPathEngineDouble::_generate1D(double *d_paths, hiprandState_t *states, FDMScheme scheme,
		unsigned int nPaths, unsigned int nSteps, double dt) const {
		// initialise RNG states
		const unsigned int threadsPerBlock = THREADS_PER_BLOCK;
		unsigned int blocksPerGrid = (nPaths + threadsPerBlock - 1) / threadsPerBlock;
		random_kernel_initializers::initialiseRandomKernel1D << <threadsPerBlock, blocksPerGrid >> >(time(0), states, nPaths);
		switch (scheme) {
		case FDMScheme::EulerScheme:
		{
			heston_kernels::euler_scheme::generatePathsKernelDouble1D << <threadsPerBlock, blocksPerGrid >> >(this->heston_, d_paths, states,
				nPaths, nSteps, dt);

		}
		break;
		case FDMScheme::MilsteinScheme:
		{
			throw std::exception("Not yet impolemented!");
		}
		break;
		}
	}

	void fdm_engine_cuda::HestonPathEngineDouble::_generate2D(double *d_paths, hiprandState_t *states, FDMScheme scheme,
		unsigned int nPaths, unsigned int nSteps, double dt)const {
		const unsigned int widthSize{ 1000 };
		assert((nPaths%widthSize) == 0);
		unsigned int heightSize{ nPaths / widthSize };
		const unsigned int threadsPerBlockX = THREADS_2D_PER_BLOCK_X;
		const unsigned int threadsPerBlockY = THREADS_2D_PER_BLOCK_Y;
		unsigned int blocksPerGridX = (widthSize + threadsPerBlockX - 1) / threadsPerBlockX;
		unsigned int blocksPerGridY = (heightSize + threadsPerBlockY - 1) / threadsPerBlockY;
		const dim3 blockSize = dim3(threadsPerBlockX, threadsPerBlockY);
		const dim3 gridSize = dim3(blocksPerGridX, blocksPerGridY);
		random_kernel_initializers::initialiseRandomKernel2D << <gridSize, blockSize >> > (time(0), states, widthSize, heightSize);
		switch (scheme) {
		case FDMScheme::EulerScheme:
		{
			heston_kernels::euler_scheme::generatePathsKernelDouble2D << <gridSize, blockSize >> > (this->heston_, d_paths, states,
				widthSize, heightSize, nSteps, dt);
		}
		break;
		case FDMScheme::MilsteinScheme:
		{
			throw std::exception("Not yet impolemented!");
		}
		break;
		}
	}


	void fdm_engine_cuda::HestonPathEngineDouble::_generate3D(double *d_paths, hiprandState_t *states, FDMScheme scheme,
		unsigned int nPaths, unsigned int nSteps, double dt)const {
		const unsigned int widthSize{ 100 };
		const unsigned int heightSize{ 100 };
		assert((nPaths % (widthSize*heightSize)) == 0);
		unsigned int depthSize{ nPaths / (widthSize*heightSize) };
		const unsigned int threadsPerBlockX = THREADS_3D_PER_BLOCK_X;
		const unsigned int threadsPerBlockY = THREADS_3D_PER_BLOCK_Y;
		const unsigned int threadsPerBlockZ = THREADS_3D_PER_BLOCK_Z;
		unsigned int blocksPerGridX = (widthSize + threadsPerBlockX - 1) / threadsPerBlockX;
		unsigned int blocksPerGridY = (heightSize + threadsPerBlockY - 1) / threadsPerBlockY;
		unsigned int blocksPerGridZ = (depthSize + threadsPerBlockZ - 1) / threadsPerBlockZ;
		const dim3 blockSize = dim3(threadsPerBlockX, threadsPerBlockY, threadsPerBlockZ);
		const dim3 gridSize = dim3(blocksPerGridX, blocksPerGridY, blocksPerGridZ);
		random_kernel_initializers::initialiseRandomKernel3D << <gridSize, blockSize >> > (time(0), states, widthSize, heightSize, depthSize);

		switch (scheme) {
		case FDMScheme::EulerScheme:
		{
			heston_kernels::euler_scheme::generatePathsKernelDouble3D << <gridSize, blockSize >> > (this->heston_, d_paths, states,
				widthSize, heightSize, depthSize, nSteps, dt);
		}
		break;
		case FDMScheme::MilsteinScheme:
		{
			throw std::exception("Not yet impolemented!");
		}
		break;
		}
	}


	fdm_engine_cuda::PathValuesType<fdm_engine_cuda::PathValuesType<double>>
		fdm_engine_cuda::HestonPathEngineDouble::simulate(unsigned int nPaths, unsigned int nSteps, double dt,
			FDMScheme scheme, GPUConfiguration config)const {

		double *d_paths = NULL;
		hiprandState_t *states; // RNG state for each thread
							   // Allocate memory for the paths
		hipMalloc(&d_paths, nPaths * nSteps * sizeof(double));
		// Allocate memory for RNG states
		hipMalloc(&states, nPaths * sizeof(hiprandState_t));

		switch (config) {
		case GPUConfiguration::Grid1D:
		{
			_generate1D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		case GPUConfiguration::Grid2D:
		{
			_generate2D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		case GPUConfiguration::Grid3D:
		{
			_generate3D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		default:
		{
			_generate1D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		}

		// Allocate memory on the host:
		double *h_paths = (double *)malloc(nPaths*nSteps * sizeof(double));
		// Copy from device to host:
		hipMemcpy(h_paths, d_paths, nPaths*nSteps * sizeof(double),
			hipMemcpyKind::hipMemcpyDeviceToHost);


		std::vector<std::vector<double>> paths(nPaths);
		for (std::size_t s = 0; s < paths.size(); ++s) {
			std::vector<double> path(nSteps);
			for (std::size_t p = 0; p < path.size(); ++p) {
				path[p] = std::move(h_paths[s + paths.size()*p]);
			}
			paths[s] = std::move(path);
		}
		free(h_paths);
		hipFree(d_paths);
		hipFree(states);
		return paths;
	}



	void fdm_engine_cuda::HestonPathEngineFloat::_generate1D(float *d_paths, hiprandState_t *states, FDMScheme scheme,
		unsigned int nPaths, unsigned int nSteps, float dt) const {
		// initialise RNG states
		const unsigned int threadsPerBlock = THREADS_PER_BLOCK;
		unsigned int blocksPerGrid = (nPaths + threadsPerBlock - 1) / threadsPerBlock;
		random_kernel_initializers::initialiseRandomKernel1D << <threadsPerBlock, blocksPerGrid >> > (time(0), states, nPaths);
		switch (scheme) {
		case FDMScheme::EulerScheme:
		{
			heston_kernels::euler_scheme::generatePathsKernelFloat1D << <threadsPerBlock, blocksPerGrid >> > (this->heston_, d_paths, states,
				nPaths, nSteps, dt);
		}
		break;
		case FDMScheme::MilsteinScheme:
		{
			throw std::exception("Not yet implementd.");
		}
		break;
		}
	}

	void fdm_engine_cuda::HestonPathEngineFloat::_generate2D(float *d_paths, hiprandState_t *states, FDMScheme scheme,
		unsigned int nPaths, unsigned int nSteps, float dt)const {
		const unsigned int widthSize{ 1000 };
		assert((nPaths%widthSize) == 0);
		unsigned int heightSize{ nPaths / widthSize };
		const unsigned int threadsPerBlockX = THREADS_2D_PER_BLOCK_X;
		const unsigned int threadsPerBlockY = THREADS_2D_PER_BLOCK_Y;
		unsigned int blocksPerGridX = (widthSize + threadsPerBlockX - 1) / threadsPerBlockX;
		unsigned int blocksPerGridY = (heightSize + threadsPerBlockY - 1) / threadsPerBlockY;
		const dim3 blockSize = dim3(threadsPerBlockX, threadsPerBlockY);
		const dim3 gridSize = dim3(blocksPerGridX, blocksPerGridY);
		random_kernel_initializers::initialiseRandomKernel2D << <gridSize, blockSize >> > (time(0), states, widthSize, heightSize);

		switch (scheme) {
		case FDMScheme::EulerScheme:
		{
			heston_kernels::euler_scheme::generatePathsKernelFloat2D << <gridSize, blockSize >> > (this->heston_, d_paths, states,
				widthSize, heightSize, nSteps, dt);
		}
		break;
		case FDMScheme::MilsteinScheme:
		{
			throw std::exception("Not yet implementd.");
		}
		break;
		}
	}

	void fdm_engine_cuda::HestonPathEngineFloat::_generate3D(float *d_paths, hiprandState_t *states, FDMScheme scheme,
		unsigned int nPaths, unsigned int nSteps, float dt)const {
		const unsigned int widthSize{ 100 };
		const unsigned int heightSize{ 100 };
		assert((nPaths % (widthSize*heightSize)) == 0);
		unsigned int depthSize{ nPaths / (widthSize*heightSize) };
		const unsigned int threadsPerBlockX = THREADS_3D_PER_BLOCK_X;
		const unsigned int threadsPerBlockY = THREADS_3D_PER_BLOCK_Y;
		const unsigned int threadsPerBlockZ = THREADS_3D_PER_BLOCK_Z;
		unsigned int blocksPerGridX = (widthSize + threadsPerBlockX - 1) / threadsPerBlockX;
		unsigned int blocksPerGridY = (heightSize + threadsPerBlockY - 1) / threadsPerBlockY;
		unsigned int blocksPerGridZ = (depthSize + threadsPerBlockZ - 1) / threadsPerBlockZ;
		const dim3 blockSize = dim3(threadsPerBlockX, threadsPerBlockY, threadsPerBlockZ);
		const dim3 gridSize = dim3(blocksPerGridX, blocksPerGridY, blocksPerGridZ);
		random_kernel_initializers::initialiseRandomKernel3D << <gridSize, blockSize >> > (time(0), states, widthSize, heightSize, depthSize);
		switch (scheme) {
		case FDMScheme::EulerScheme:
		{
			heston_kernels::euler_scheme::generatePathsKernelFloat3D << <gridSize, blockSize >> > (this->heston_, d_paths, states,
				widthSize, heightSize, depthSize, nSteps, dt);
		}
		break;
		case FDMScheme::MilsteinScheme:
		{
			throw std::exception("Not yet implementd.");
		}
		break;
		}
	}


	fdm_engine_cuda::PathValuesType<fdm_engine_cuda::PathValuesType<float>>
		fdm_engine_cuda::HestonPathEngineFloat::simulate(unsigned int nPaths, unsigned int nSteps, float dt,
			FDMScheme scheme, GPUConfiguration config)const {

		float *d_paths = NULL;
		hiprandState_t *states; // RNG state for each thread
							   // Allocate memory for the paths
		hipMalloc(&d_paths, nPaths * nSteps * sizeof(float));
		// Allocate memory for RNG states
		hipMalloc(&states, nPaths * sizeof(hiprandState_t));


		switch (config) {
		case GPUConfiguration::Grid1D:
		{
			_generate1D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		case GPUConfiguration::Grid2D:
		{
			_generate2D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		case GPUConfiguration::Grid3D:
		{
			_generate3D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		default:
		{
			_generate1D(d_paths, states, scheme, nPaths, nSteps, dt);
		}
		break;
		}

		// Allocate memory on the host:
		float *h_paths = (float *)malloc(nPaths*nSteps * sizeof(float));
		// Copy from device to host:
		hipMemcpy(h_paths, d_paths, nPaths*nSteps * sizeof(float),
			hipMemcpyKind::hipMemcpyDeviceToHost);

		std::vector<std::vector<float>> paths(nPaths);
		for (std::size_t s = 0; s < paths.size(); ++s) {
			std::vector<float> path(nSteps);
			for (std::size_t p = 0; p < path.size(); ++p) {
				path[p] = std::move(h_paths[s + paths.size()*p]);
			}
			paths[s] = std::move(path);
		}


		free(h_paths);
		hipFree(d_paths);
		hipFree(states);

		return paths;
	}



}


